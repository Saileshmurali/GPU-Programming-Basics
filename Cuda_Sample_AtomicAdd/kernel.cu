#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


__global__ void AtomicAddCheck(int* d_pointer)
{
	int idx = threadIdx.x;
	__shared__ int val;
	val = 0;
	if (idx % 2 == 0)
	{
		atomicAdd(&val, 1);
		*d_pointer = val;
	}
	else
	{
		atomicAdd(&val, 2);
		*d_pointer = val;
	}

}

int main()
{
	int h_var = 0, h_threads = 100;
	int* h_pointer = &h_var;
	int* d_pointer;
	hipMalloc(&d_pointer, sizeof(int));
	hipMemcpy(d_pointer, h_pointer, sizeof(int), hipMemcpyHostToDevice);

	AtomicAddCheck << <1, 100 >> > (d_pointer);
	hipDeviceSynchronize();
	hipMemcpy(h_pointer, d_pointer, sizeof(int), hipMemcpyDeviceToHost);
	printf("The sum is %d", *h_pointer);
}