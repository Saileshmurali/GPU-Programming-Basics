﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void Square(int* GPU_IN, int* GPU_OUT)
{
	int idx = threadIdx.x;
	int n = GPU_IN[idx];
	GPU_OUT[idx] = n * n;
	//printf("%d\n", GPU_OUT[idx]);
}

int main()
{
	const int Array_Length = 21;
	const int Array_Size = Array_Length * sizeof(int);

	int* CPU_IN = (int *)malloc(Array_Size);
	int* CPU_OUT = (int*)malloc(Array_Size);
	for (int i = 0;i < Array_Length;i++)
	{
		CPU_IN[i] = i;
		//CPU_OUT[i] = 0;
		//printf("%d\n", CPU_IN[i]);
	}
	int* GPU_IN, * GPU_OUT;
	hipMalloc(& GPU_IN, Array_Size);
	hipMalloc(& GPU_OUT, Array_Size);
	hipMemcpy(GPU_IN, CPU_IN, Array_Size, hipMemcpyHostToDevice);

	Square <<<1, Array_Length >>> (GPU_IN, GPU_OUT);
	hipDeviceSynchronize();
	hipMemcpy(CPU_OUT, GPU_OUT, Array_Size, hipMemcpyDeviceToHost);
	for (int i = 0;i < Array_Length;i++)
	{
		printf("%d\n", CPU_OUT[i]);
	}
	free(CPU_IN);
	free(CPU_OUT);
	hipFree(GPU_IN);
	hipFree(GPU_OUT);
}